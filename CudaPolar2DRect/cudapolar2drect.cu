#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <random>
#include <chrono>
#define _USE_MATH_DEFINES
#include <math.h>

using namespace std;

#define Du 0.2
#define chi 8
#define au 1
#define Bv 0.73
#define R 60
#define F 200
#define dt 0.00005
const double H_dr = 9 / (M_PI * R); /* 2pi(R*dr) = 360*dx (cartesian sim) */
const double H_dr2 = H_dr * H_dr;
const double H_BASE_df = 2 * M_PI / F;

__constant__ double dr;
__constant__ double hdr;
__constant__ double dr2;
__constant__ double BASE_df;

#define FRAME_DURATION 6
//#define H 128
//#define L 3456000
#define L 3456000
#define SNAPSHOT_STEP 3600


#define BITMASK4 0xFFFFFFFC
#define BITMASK2 0xFFFFFFFE

// fp - f+1
// fm - f-1
// rp - r+1
// rm - r-1
__device__
inline double getNextU(
	double u, double urp, double urm, double ufp, double ufm,
	double v, double vrp, double vrm, double vfp, double vfm,
	double ro, double df
)
{
	double urp2 = (u + urp) / 2;
	double urm2 = (u + urm) / 2;
	double ufp2 = (u + ufp) / 2;
	double ufm2 = (u + ufm) / 2;
	double rp2 = ro + hdr;
	double rm2 = ro - hdr;

	return dt * (
		Du * (
			(rp2 * (urp - u) - rm2 * (u - urm)) / (ro * dr2)
			+ (ufp - 2 * u + ufm) / (ro * ro * df * df)
			) - chi * (
				(rp2 * urp2 * (vrp - v) - rm2 * urm2 * (v - vrm)) / (ro * dr2)
				+ (ufp2 * (vfp - v) - ufm2 * (v - vfm)) / (ro * ro * df * df)
				) + au * u * (1 - u)
		) + u;
}

__device__
inline double getNextV(
	double u,
	double v, double vrp, double vrm, double vfp, double vfm,
	double ro, double df
)
{
	double rp2 = ro + hdr;
	double rm2 = ro - hdr;

	return dt * (
		(rp2 * (vrp - v) - rm2 * (v - vrm)) / (ro * dr2)
		+ (vfp - 2 * v + vfm) / (ro * ro * df * df)
		+ u / (1 + Bv * u) - v
		) + v;
}

__global__
void boundaryKernel(double* u, double* v)
{
	int fOffset = threadIdx.x + blockIdx.x * blockDim.x;
	int fStride = blockDim.x * gridDim.x;

	for (int f = fOffset; f < F / 2; f += fStride)
	{
		// no-flux boundary condition
		u[(R - 1) * F + f] = (4 * u[(R - 2) * F + f] - u[(R - 3) * F + f]) / 3;
		v[(R - 1) * F + f] = (4 * v[(R - 2) * F + f] - v[(R - 3) * F + f]) / 3;
		// central symmetry boundary condition
		if (f % 4 == 0) {
			u[F / 2 + f] = u[f] = (u[F + f] + u[3 * F / 2 + f]) / 2;
			v[F / 2 + f] = v[f] = (v[F + f] + v[3 * F / 2 + f]) / 2;
		}
	}
	for (int f = fOffset + F / 2; f < F; f += fStride)
	{
		// no-flux boundary condition
		u[(R - 1) * F + f] = (4 * u[(R - 2) * F + f] - u[(R - 3) * F + f]) / 3;
		v[(R - 1) * F + f] = (4 * v[(R - 2) * F + f] - v[(R - 3) * F + f]) / 3;
	}
}

__global__
void calcKernel(double* uOutput, double* vOutput, double* uInput, double* vInput)
{
	int rOffset = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int rStride = blockDim.x * gridDim.x;
	int fOffset = threadIdx.y + blockIdx.y * blockDim.y;
	int fStride = blockDim.y * gridDim.y;

	double urp, urm, vrp, vrm, ufp, ufm, vfp, vfm, u, v;
	double df, ro;
	int fp, fm;

	for (int r = rOffset; r < R - 1; r += rStride)
	{
		ro = r * dr;
		for (int f = fOffset; f < F; f += fStride)
		{
			if (r < R / 4 && f % 4 != 0)
				continue;
			if (r < R / 2 && f % 2 != 0)
				continue;

			u = uInput[F * r + f];
			v = vInput[F * r + f];

			if (r == R / 4 - 1) {
				urp = (uInput[F * (r + 1) + f] + uInput[F * (r + 1) + f + 2]) / 2;
				vrp = (vInput[F * (r + 1) + f] + vInput[F * (r + 1) + f + 2]) / 2;
			}
			else if (r == R / 2 - 1) {
				urp = (uInput[F * (r + 1) + f] + uInput[F * (r + 1) + f + 1]) / 2;
				vrp = (vInput[F * (r + 1) + f] + vInput[F * (r + 1) + f + 1]) / 2;
			}
			else {
				urp = uInput[F * (r + 1) + f];
				vrp = vInput[F * (r + 1) + f];
			}

			if (r == R / 4) {
				urm = uInput[F * (r - 1) + (f & BITMASK4)] / 2;
				vrm = vInput[F * (r - 1) + (f & BITMASK4)] / 2;
			}
			else if (r == R / 2) {
				urm = uInput[F * (r - 1) + (f & BITMASK2)] / 2;
				vrm = vInput[F * (r - 1) + (f & BITMASK2)] / 2;
			}
			else {
				urm = uInput[F * (r - 1) + f];
				vrm = vInput[F * (r - 1) + f];
			}

			if (r < R / 4)
				fp = f + 4;
			else if (r < R / 2)
				fp = f + 2;
			else
				fp = f + 1;
			if (fp >= F) fp = 0;
			ufp = uInput[F * r + fp];
			vfp = vInput[F * r + fp];

			if (r < R / 4) {
				fm = f - 4;
			}
			else if (r < R / 2) {
				fm = f - 2;
			}
			else {
				fm = f - 1;
			}
			if (fm < 0) fm += F;
			ufm = uInput[F * r + fm];
			vfm = vInput[F * r + fm];

			if (r < R / 4)
				df = BASE_df * 4;
			else if (r < R / 2)
				df = BASE_df * 2;
			else
				df = BASE_df;

			uOutput[F * r + f] = getNextU(u, urp, urm, ufp, ufm, v, vrp, vrm, vfp, vfm, ro, df);
			vOutput[F * r + f] = getNextV(u, v, vrp, vrm, vfp, vfm, ro, df);
		}
	}
}

int main()
{
	double* matrixU1, * matrixU2, * matrixV1, * matrixV2;
	int bufferlength = R * F;
	int size = bufferlength * sizeof(double);
	double H_hdr = H_dr / 2;
	hipError_t cudaErr;

	cudaErr = hipMemcpyToSymbol(HIP_SYMBOL(dr), &H_dr, sizeof(double));
	if (cudaErr != hipSuccess)
		cout << "hipMemcpyToSymbol: " << hipGetErrorString(cudaErr) << endl;
	cudaErr = hipMemcpyToSymbol(HIP_SYMBOL(dr2), &H_dr2, sizeof(double));
	if (cudaErr != hipSuccess)
		cout << "hipMemcpyToSymbol: " << hipGetErrorString(cudaErr) << endl;
	cudaErr = hipMemcpyToSymbol(HIP_SYMBOL(BASE_df), &H_BASE_df, sizeof(double));
	if (cudaErr != hipSuccess)
		cout << "hipMemcpyToSymbol: " << hipGetErrorString(cudaErr) << endl;
	cudaErr = hipMemcpyToSymbol(HIP_SYMBOL(hdr), &H_hdr, sizeof(double));
	if (cudaErr != hipSuccess)
		cout << "hipMemcpyToSymbol: " << hipGetErrorString(cudaErr) << endl;

	double* matrixUInit = new double[bufferlength];
	double* matrixVInit = new double[bufferlength];

	cudaErr = hipMalloc(&matrixU1, size);
	if (cudaErr != hipSuccess)
		cout << "hipMalloc: " << hipGetErrorString(cudaErr) << endl;
	cudaErr = hipMalloc(&matrixU2, size);
	if (cudaErr != hipSuccess)
		cout << "hipMalloc: " << hipGetErrorString(cudaErr) << endl;
	cudaErr = hipMalloc(&matrixV1, size);
	if (cudaErr != hipSuccess)
		cout << "hipMalloc: " << hipGetErrorString(cudaErr) << endl;
	cudaErr = hipMalloc(&matrixV2, size);
	if (cudaErr != hipSuccess)
		cout << "hipMalloc: " << hipGetErrorString(cudaErr) << endl;

	int deviceId;
	cudaErr = hipGetDevice(&deviceId);
	if (cudaErr != hipSuccess)
		cout << "hipGetDevice: " << hipGetErrorString(cudaErr) << endl;

	cudaErr = hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	if (cudaErr != hipSuccess)
		cout << "hipDeviceSetCacheConfig: " << hipGetErrorString(cudaErr) << endl;

	cout << size * (L / SNAPSHOT_STEP + 1) << endl;
	double* matrixU = (double*)malloc(size * (L / SNAPSHOT_STEP + 1)); // for gif output
	perror("malloc error");
	double* matrixV = (double*)malloc(size * (L / SNAPSHOT_STEP + 1));
	perror("malloc error");
	auto seed = chrono::system_clock::now().time_since_epoch().count();
	normal_distribution<double> distr(0, 0.1);
	default_random_engine re(1);

	for (int i = 0; i < bufferlength; i++)
	{
		matrixU[i] = matrixUInit[i] = distr(re) + 1.0;
		matrixV[i] = matrixVInit[i] = 0;
	}

	hipMemcpy(matrixU1, matrixUInit, size, hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(matrixV1, matrixVInit, size, hipMemcpyKind::hipMemcpyHostToDevice);
	delete matrixUInit;
	delete matrixVInit;

	dim3 blocks(1, 10);
	dim3 threads(32, 20);

	hipGetLastError();
	auto start = clock();
	double* temp = NULL;
	for (int i = 0; i < L; ++i)
	{
		if (i % SNAPSHOT_STEP == SNAPSHOT_STEP - 1)
		{
			hipDeviceSynchronize();
			int step = i / SNAPSHOT_STEP + 1;
			//save frame
			hipMemcpy(matrixU + step * bufferlength, matrixU1, size, hipMemcpyKind::hipMemcpyDeviceToHost);
			hipMemcpy(matrixV + step * bufferlength, matrixV1, size, hipMemcpyKind::hipMemcpyDeviceToHost);
			double elapsed = (clock() - start) / (double)CLOCKS_PER_SEC;
			cout << "step " << step << ", time elapsed: " << elapsed << ", avg: " << elapsed / step << endl;
		}

		calcKernel <<< blocks, threads >>> (matrixU2, matrixV2, matrixU1, matrixV1);
		cudaErr = hipGetLastError();
		if (cudaErr != hipSuccess)
			cout << "calcKernel: " << hipGetErrorString(cudaErr) << endl;


		boundaryKernel <<< 10, 64 >>> (matrixU2, matrixV2);
		cudaErr = hipGetLastError();
		if (cudaErr != hipSuccess)
			cout << "boundaryKernel: " << hipGetErrorString(cudaErr) << endl;


		// pointer swap
		temp = matrixU1;
		matrixU1 = matrixU2;
		matrixU2 = temp;

		temp = matrixV1;
		matrixV1 = matrixV2;
		matrixV2 = temp;
	}
	auto duration = (clock() - start) / (double)CLOCKS_PER_SEC;
	cout << "duration: " << duration << endl;

	ofstream datustream;
	datustream.open("u.dat", ios::binary | ios::out);
	if (datustream.is_open())
	{
		cout << "stream open!" << endl;
		datustream.write((char*)matrixU, size * (L / SNAPSHOT_STEP + 1));
		datustream.close();
	}

	ofstream datvstream;
	datvstream.open("v.dat", ios::binary | ios::out);
	if (datvstream.is_open())
	{
		cout << "stream open!" << endl;
		datvstream.write((char*)matrixV, size * (L / SNAPSHOT_STEP + 1));
		datvstream.close();
	}

	FILE* csvu, * csvv;
	errno_t err = fopen_s(&csvu, "u.csv", "w");
	if (err) return err;
	err = fopen_s(&csvv, "v.csv", "w");
	if (err) return err;
	for (int j = 0; j <= L / SNAPSHOT_STEP; j++)
	{
		for (int i = 0; i < bufferlength; i+=4)
		{
			fprintf(csvu, "%f;", matrixU[j * bufferlength + i]);
			fprintf(csvv, "%f;", matrixV[j * bufferlength + i]);
		}
		fprintf(csvu, "\n");
		fprintf(csvv, "\n");
	}

	fclose(csvu);
	fclose(csvv);
}