#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <gd.h>
#include <random>
#include <chrono>
#include "Constants.h"
#include "../RectPNG/RectPNG.h"
#include "argh.h"
#include "../Polar2DShared/UV.cpp"
#include "../PolarPNG/PolarPNG.h"

#define GET(arr, x, y, z) (arr[(x) + (y) * X + (z) * XY])

using namespace std;

#define L 300000
#define SNAPSHOT_STEP 100000

__global__
void calcKernel(double* uOutput, double* vOutput, double* uInput, double* vInput)
{
	int xOffset = threadIdx.x + blockIdx.x * blockDim.x;
	int xStride = blockDim.x * gridDim.x;

	for (int r = 1; r < R / 4; ++r) {
		double ro = r * dr;
		for (int f = xOffset * 4; f < F; f += xStride * 4) {
			calcPoint(uOutput, vOutput, uInput, vInput, ro, r, f);
		}
	}

	for (int r = R / 4; r < R / 2; ++r) {
		double ro = r * dr;
		for (int f = xOffset * 2; f < F; f += xStride * 2) {
			calcPoint(uOutput, vOutput, uInput, vInput, ro, r, f);
		}
	}

	for (int r = R / 2; r < R - 1; ++r) {
		double ro = r * dr;
		for (int f = xOffset; f < F; f += xStride) {
			calcPoint(uOutput, vOutput, uInput, vInput, ro, r, f);
		}
	}
}

// apply boundary conditions
__global__
void boundaryKernel(double* u, double* v)
{
	int xOffset = threadIdx.x + blockIdx.x * blockDim.x;
	int xStride = blockDim.x * gridDim.x;
	
	calcBoundary(u, v, xOffset, xStride);
}

int main(int argc, char* argv[])
{
	initializeCudaConstants();
	hipError_t err;

	int bufferlength = R * F;
	double* matrixU1, * matrixU2, * matrixV1, * matrixV2;
	int size = bufferlength * sizeof(double);
	err = hipMalloc(&matrixU1, size);
	if (err != hipSuccess)
		cout << "hipMalloc: " << hipGetErrorString(err) << endl;
	err = hipMalloc(&matrixU2, size);
	if (err != hipSuccess)
		cout << "hipMalloc: " << hipGetErrorString(err) << endl;
	err = hipMalloc(&matrixV1, size);
	if (err != hipSuccess)
		cout << "hipMalloc: " << hipGetErrorString(err) << endl;
	err = hipMalloc(&matrixV2, size);
	if (err != hipSuccess)
		cout << "hipMalloc: " << hipGetErrorString(err) << endl;
	int deviceId;
	err = hipGetDevice(&deviceId);
	if (err != hipSuccess)
		cout << "hipGetDevice: " << hipGetErrorString(err) << endl;

	double* matrixU = new double[bufferlength];
	double* matrixV = new double[bufferlength];

	cout << size * (L / SNAPSHOT_STEP + 1) << endl;
	auto seed = chrono::system_clock::now().time_since_epoch().count();
	normal_distribution<double> distr(0, 0.1);
	cout << seed << endl;
	default_random_engine re(1);
	for (int i = 0; i < bufferlength; i++)
	{
		if (i < bufferlength / 2 && i % 2 != 0 || i < bufferlength / 4 && i % 4 != 0)
			matrixU[i] = 0.0;
		else
			matrixU[i] = distr(re) + 1.0;
		matrixV[i] = 0.0;
	}

	hipMemcpy(matrixU1, matrixU, size, hipMemcpyHostToDevice);
	hipMemcpy(matrixV1, matrixV, size, hipMemcpyHostToDevice);
	hipMemcpy(matrixU2, matrixU, size, hipMemcpyHostToDevice);
	hipMemcpy(matrixV2, matrixV, size, hipMemcpyHostToDevice);
	boundaryKernel <<< 28, 8 >>> (matrixU1, matrixV1);

	PolarPNG uPng(R, 3, F, 4.5);
	PolarPNG vPng(R, 3, F, 0.6);

	uPng.savePNG(matrixU, "u_step0.png");
	vPng.savePNG(matrixV, "v_step0.png");

	ofstream datustream;
	datustream.open("u.dat", ios::binary | ios::out);
	ofstream datvstream;
	datvstream.open("v.dat", ios::binary | ios::out);

	if (datustream.is_open())
		datustream.write((char*)matrixU, size);
	if (datvstream.is_open())
		datvstream.write((char*)matrixV, size);

#ifdef _DEBUG
	int warpSize, multiProcessorCount;
	hipDeviceProp_t props;
	hipDeviceGetAttribute(&warpSize, hipDeviceAttribute_t::hipDeviceAttributeWarpSize, deviceId);
	hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttribute_t::hipDeviceAttributeMultiprocessorCount, deviceId);
	hipGetDeviceProperties(&props, deviceId);
#endif

	auto start = clock();
	auto start_current = start;
	double* temp = nullptr;
	for (int i = 0; i < L; i++)
	{
		hipGetLastError(); // flush previous errors
		calcKernel <<< 7, 32 >>> (matrixU2, matrixV2, matrixU1, matrixV1);
		err = hipGetLastError();
		if (err != hipSuccess)
			cout << "calcKernel: " << hipGetErrorString(err) << endl;

		boundaryKernel <<< 7, 32 >>> (matrixU2, matrixV2);
		err = hipGetLastError();
		if (err != hipSuccess)
			cout << "boundaryKernel: " << hipGetErrorString(err) << endl;

		if (i % SNAPSHOT_STEP == SNAPSHOT_STEP - 1)
		{
			hipDeviceSynchronize();
			int step = i / SNAPSHOT_STEP + 1;

			// save frame
			clock_t saveframe = clock();

			hipGetLastError(); // flush previous errors
			hipMemcpy(matrixU, matrixU2, size, hipMemcpyDeviceToHost);
			err = hipGetLastError();
			if (err != hipSuccess)
				cout << "hipMemcpy: " << hipGetErrorString(err) << endl;
			hipMemcpy(matrixV, matrixV2, size, hipMemcpyDeviceToHost);
			err = hipGetLastError();
			if (err != hipSuccess)
				cout << "hipMemcpy: " << hipGetErrorString(err) << endl;

			if (datustream.is_open())
				datustream.write((char*)matrixU, size);
			if (datvstream.is_open())
				datvstream.write((char*)matrixV, size);

			uPng.savePNG(matrixU, "u_step" + to_string(step) + ".png");
			vPng.savePNG(matrixV, "v_step" + to_string(step) + ".png");
			double done = clock();

			double processedIn = (saveframe - start_current) / (double)CLOCKS_PER_SEC;
			double outputIn = (done - saveframe) / (double)CLOCKS_PER_SEC;
			double totalTime = (done - start) / (double)CLOCKS_PER_SEC;
			cout << "step " << step << ", done processing in: " << processedIn << ", saved snapshot in: " << outputIn << ", total: " << totalTime << ", avg: " << totalTime / step << endl;
			start_current = clock();
		}

		// pointer swap
		temp = matrixU1;
		matrixU1 = matrixU2;
		matrixU2 = temp;

		temp = matrixV1;
		matrixV1 = matrixV2;
		matrixV2 = temp;
	}
	auto duration = (clock() - start) / (double)CLOCKS_PER_SEC;
	cout << "duration: " << duration << endl;

	delete[] matrixU, matrixV;
	hipFree(matrixU1);
	hipFree(matrixU2);
	hipFree(matrixV1);
	hipFree(matrixV2);
}
